#include "hip/hip_runtime.h"
// nvcc -arch=sm_35 -std=c++17 -lnvidia-ml  ./benchmark.cu ../../depends/libff-cuda/curves/bls12_381/bls12_381_pp_host.cu ../../depends/libff-cuda/curves/bls12_381/bls12_381_init_host.cu  -o benchmark && ./benchmark 25

//#include "../../depends/libff-cuda/curves/bls12_381/bls12_381_init_host.cuh"
//#include "../../depends/libff-cuda/curves/bls12_381/bls12_381_g1_host.cuh"
//#include "../../depends/libff-cuda/curves/bls12_381/bls12_381_g2_host.cuh"
//#include "../../depends/libff-cuda/fields/bigint_host.cuh"
//#include "../../depends/libff-cuda/fields/fp_host.cuh"
#include "../../depends/libff-cuda/curves/bls12_381/bls12_381_pp_host.cuh"
//#include "../../depends/libff-cuda/curves/bls12_381/bls12_381_init_host.cuh"
#include "../../depends/libff-cuda/curves/bls12_381/bls12_381_init.cuh"
#include "../../depends/libff-cuda/curves/bls12_381/bls12_381_pp.cuh"
//#include "../../depends/libff-cuda/fields/bigint_host.cuh"
using namespace libff;

//#include "../../depends/libstl-cuda/memory.cuh"
//#include "../../depends/libstl-cuda/vector.cuh"

#include <iostream>
#include <nvml.h>
using namespace std;


template<typename ppT>
struct MSM_params
{
    libstl::vector<libff::Fr<ppT>> vf;
    libstl::vector<libff::G1<ppT>> vg;
};

struct instance_params
{
    bls12_381_Fr instance;
    bls12_381_G1 g1_instance;
    bls12_381_G2 g2_instance;
    bls12_381_GT gt_instance;
};

struct h_instance_params
{
    bls12_381_Fr_host h_instance;
    bls12_381_G1_host h_g1_instance;
    bls12_381_G2_host h_g2_instance;
    bls12_381_GT_host h_gt_instance;
};



#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(const char* const file, const int line)
{
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << endl;
        cerr << hipGetErrorString(err) << endl;
    }
}

__global__ void generate_MP(libstl::vector<Fr<bls12_381_pp>>& vf, libstl::vector<G1<bls12_381_pp>>& vg, size_t size)
{
    for (int i=0; i<size; i++) {
        vf[i]  = libff::bls12_381_Fr(&bls12_381_fp_params_r);
        vg[i]  = libff::bls12_381_G1(&g1_params);
    };
}

int main(int argc, char* argv[])
{
    // initialize NVML objects for CUDA profiling
    nvmlReturn_t status;
    nvmlDevice_t device_handle;
    unsigned long long energy_start, energy_end;
    long long energy_total;
    hipEvent_t time_start, time_stop;
    float time_total;
    unsigned int memClock;
    unsigned int ClockFreqNumber=200;
    unsigned int ClockFreqs[200];
    hipEventCreate(&time_start);
    hipEventCreate(&time_stop);
    nvmlInit();
    nvmlDeviceGetHandleByIndex(0, &device_handle);
    nvmlDeviceGetApplicationsClock(device_handle, NVML_CLOCK_MEM, &memClock);
    nvmlDeviceGetSupportedGraphicsClocks(device_handle, memClock, &ClockFreqNumber, ClockFreqs );

    // prepare data
    bls12_381_pp_host::init_public_params();
    //libff::Fr<bls12_381_pp> f = bls12_381_Fr.random_element();
    //libff::G1<bls12_381_pp> g = bls12_381_G1.random_element();
    //MSM_params<bls12_381_pp> mp;
    MSM_params<bls12_381_pp> mp;
    printf("Size scalar+point=%d",sizeof(mp));
    printf("Size scalar=%d,point=%d",sizeof(libff::Fr<bls12_381_pp>),sizeof(libff::G1<bls12_381_pp>));

    libstl::vector<Fr<bls12_381_pp>> mp_vf;
    libstl::vector<G1<bls12_381_pp>> mp_vg;
    generate_MP<<<1,1>>>(mp_vf,mp_vg, 10);
    //hipMalloc( (void**)&mp[i], sizeof(MSM_params<bls12_381_pp>))

    //MSM_params<bls12_381_pp>* mp;
    //hipMalloc( (void**)&mp, sizeof(MSM_params<bls12_381_pp>));
    CHECK_LAST_CUDA_ERROR();
    printf("Dupa\n");
    nvmlShutdown();
}